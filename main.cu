
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 100

#define TYPE_AND 0
#define TYPE_AVA 1

#define cudaCheckError()                                                                       \
    {                                                                                          \
        hipError_t e = hipGetLastError();                                                    \
        if (e != hipSuccess)                                                                  \
        {                                                                                      \
            printf("\nCuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        }                                                                                      \
    }

struct Node;
__device__ Node *nodes;

__device__ Node *firstAva;
__device__ Node *actualAva;

struct Node
{
    char actualIndex;
    char type;
    bool alreadyUsed = false;
    float lastValue;

    //Posso substituir por ints ou shorts, já que nodes é uma memória contígua
    Node *childs[4];
    Node *calledBy;
    Node *parent;

    float weights[4];

    __device__ int id()
    {
        return (int)(this - nodes);
    }
};

__global__ void allocNodes()
{
    hipMalloc(&nodes, sizeof(Node) * SIZE);
}

__global__ void initializeRandom()
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    hiprandState_t state;
    hiprand_init(clock64(), i * blockDim.x + j, 0, &state);

    nodes[i].childs[j] = &nodes[hiprand(&state) % SIZE];
    // printf("nodes[%d].child[%d] = %d\n", i, j, nodes[i].childs[j]->id());

    if (j == 0)
    {
        nodes[i].calledBy = &nodes[hiprand(&state) % SIZE]; //impede receive para algo fora da memoria
        nodes[i].parent = &nodes[hiprand(&state) % SIZE];   //TESTING
        // printf("%d->Node[%d]\n", nodes[i].parent->id(), nodes[i].id());
    }

    nodes[i].lastValue = i;  //marcador, exemplo: Node[3].lastvalue = 3
    nodes[i].weights[j] = i; //marcador, exemplo: Node[3].weight[i] = 3

    nodes[i].alreadyUsed = false; //por segurança
    nodes[i].actualIndex = 0;

    nodes[i].type = (i + 1) % 5 == 0 ? TYPE_AVA : TYPE_AND; //para testes iniciais

    firstAva = &nodes[4];
    actualAva = &nodes[4];
}

__global__ void descendent(Node *parent, Node *parallelParent, int limit = 0)
{
    if (parent->alreadyUsed == true || limit >= 1)
        return;

    printf("Parent Node[%d] (%d, %d, %d, %d)\n",
           parent->id(),
           parent->childs[0]->id(),
           parent->childs[1]->id(),
           parent->childs[2]->id(),
           parent->childs[3]->id());

    parent->alreadyUsed = true;

    hiprandState_t state;
    hiprand_init(clock64(), 0, 0, &state);
    // Copy in parallel parent
    for (int i = 0; i < 4; i++)
    {
        parallelParent->childs[i] = &nodes[hiprand(&state) % SIZE];

        // And copy parent->childs[i] status ..
        for (int j = 0; j < 4; j++)
        {
            parallelParent->childs[i]->weights[j] = parent->childs[i]->weights[j];
        }
    }
    printf("Parallel parent Node[%d] (%d, %d, %d, %d)\n",
           parallelParent->id(),
           parallelParent->childs[0]->id(),
           parallelParent->childs[1]->id(),
           parallelParent->childs[2]->id(),
           parallelParent->childs[3]->id());

    // Iterate next steps
    for (int i = 0; i < 4; i++)
    {
        parallelParent->childs[i] = &nodes[hiprand(&state) % SIZE];
        descendent<<<1, 1>>>(parent->childs[i], parallelParent->childs[i], limit + 1);
        __syncthreads();
    }
}

__global__ void reproduce()
{
    hiprandState_t state;
    hiprand_init(clock64(), 0, 0, &state);
    Node *parallelParent = &nodes[hiprand(&state) % SIZE];
    descendent<<<1, 1>>>(nodes, parallelParent);
}

__device__ void addInAvaList(Node *newAva)
{
    actualAva->childs[1] = newAva;
    actualAva = newAva;
}

// setter set node
__device__ bool canSet(Node *setter, Node *node)
{
    // Posso adicionar mais e mais coisa como node->parent->parent == setter->parent
    // a depender de ajuste fino para manter a cadeia
    return (
        node->parent == setter ||
        node->parent == setter->parent ||
        node->parent->parent == setter ||
        node->parent->parent == setter->parent);
}

__global__ void call(Node *caller);

__global__ void receive(Node *receiver, float value, Node *from)
{
    receiver->actualIndex++;

    // printf("Node[%d] recebeu valor %.0f\n", receiver->id(), value);

    switch (receiver->type)
    {
    //Aqui tem maior ou igual porquw actualIndex varia bastante
    case TYPE_AND:
        if (receiver->actualIndex >= 4)
        {
            receiver->actualIndex = 0;
            receive<<<1, 1>>>(receiver->calledBy, value, receiver);
            // printf("Node[%d] (and) recebeu quatro vezes\n", receiver->id());
        }
        receiver->lastValue = value;
        break;
    case TYPE_AVA:
        // printf("Node[%d] eh do tipo AVA, actualIndex: %d\n", receiver->id(), receiver->actualIndex);
        if (receiver->actualIndex == 1) // Primeira vez nessa rodada
        {
            receiver->actualIndex = 2;
            receiver->childs[0] = from;
            addInAvaList(receiver);
            receiver->lastValue = value;
            printf("Node[%d] (ava) de Node[%d] pela primeira vez na rodada, lastvalue %d\n",
                   receiver->id(), from->id(), receiver->childs[0]->id(), (int)receiver->lastValue);
        }
        else
        {
            receiver->actualIndex = 1;
        }
        receive<<<1, 1>>>(receiver->calledBy, value, receiver);
        // printf("Node[%d] (ava) recebeu uma vezes\n", receiver->id());
        break;
    }

    receiver->alreadyUsed = false; //allow node to be called several times
    // printf("Node[%d] recebe de Node[%d], lastValue: %d\n", receiver->id(), from->id(), (int)receiver->lastValue);
}

__global__ void call(Node *caller)
{
    caller->alreadyUsed = true;

    int numberOfCallChildren = caller->type == TYPE_AVA ? 1 : 4;

    for (int i = 0; i < numberOfCallChildren; i++)
    {
        if (!caller->childs[i]->alreadyUsed)
        {
            caller->childs[i]->alreadyUsed = true;
            caller->childs[i]->calledBy = caller;
            call<<<1, 1>>>(caller->childs[i]);
            // printf("Node[%d], type %d, call Node[%d]\n", caller->id(), caller->type, caller->childs[i]->id());
        }
        else
        {
            receive<<<1, 1>>>(caller, caller->childs[i]->lastValue, caller->childs[i]);
            // printf("Node[%d] tentou chamar Node[%d] usado, recebe %.0f\n", caller->id(), caller->childs[i]->id(), caller->childs[i]->lastValue);
        }
    }

    // cudaCheckError();
}

__global__ void callFirst()
{
    call<<<1, 1>>>(nodes);
}

//NESCESSARY!
__global__ void resetAlreadyUseds()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    nodes[i].alreadyUsed = false;
}

__global__ void logNodesLastValues()
{
    for (int i = 0; i < SIZE; i++)
    {
        printf("%d ", (int)nodes[i].weights[0]);
    }
}

__global__ void logAvaList()
{
    Node *trackNode = firstAva;
    printf("type firstAva: %d\n", firstAva->type);
    for (int i = 0; i < 20; i++)
    {
        if (trackNode->type != TYPE_AVA || trackNode->alreadyUsed)
            break;
        printf("->Node[%d] lastValue: %d\n", trackNode->id(), (int)trackNode->lastValue);
        trackNode->alreadyUsed = true;
        trackNode = trackNode->childs[1];
    }

    trackNode = firstAva;
    for (int i = 0; i < 20; i++)
    {
        if (trackNode->type != TYPE_AVA || !trackNode->alreadyUsed)
            break;
        trackNode->alreadyUsed = false;
        trackNode = trackNode->childs[1];
    }
}

int main()
{
    allocNodes<<<1, 1>>>();
    hipDeviceSynchronize();
    initializeRandom<<<SIZE, 4>>>();
    hipDeviceSynchronize();
    for (int i = 0; i < 1; i++)
    {
        callFirst<<<1, 1>>>();
        hipDeviceSynchronize();
        resetAlreadyUseds<<<SIZE, 1>>>();
        hipDeviceSynchronize();
        logAvaList<<<1, 1>>>();
        hipDeviceSynchronize();
        // reproduce<<<1, 1>>>();
        // cudaDeviceSynchronize();
        // logNodesLastValues<<<1, 1>>>();
        // cudaDeviceSynchronize();
    }

    cudaCheckError();
}

//UriSE tenha DETERMINAÇÃO